
#include <hip/hip_runtime.h>
/*! \file CTU_3D_cuda.cu
 *  \brief Definitions of the cuda 3D CTU algorithm functions. */

#ifdef CUDA

#include<stdio.h>
#include<stdlib.h>
#include<math.h>
#include<cuda.h>
#include"global.h"
#include"global_cuda.h"
#include"hydro_cuda.h"
#include"CTU_3D_cuda.h"
#include"pcm_cuda.h"
#include"plmp_cuda.h"
#include"plmc_cuda.h"
#include"ppmp_cuda.h"
#include"ppmc_cuda.h"
#include"exact_cuda.h"
#include"roe_cuda.h"
#include"hllc_cuda.h"
#include"h_correction_3D_cuda.h"
#include"cooling_cuda.h"
#include"subgrid_routines_3D.h"
#include"io.h"


__global__ void Evolve_Interface_States_3D(Real *dev_conserved, Real *dev_Q_Lx, Real *dev_Q_Rx, Real *dev_F_x,
                                           Real *dev_Q_Ly, Real *dev_Q_Ry, Real *dev_F_y,
                                           Real *dev_Q_Lz, Real *dev_Q_Rz, Real *dev_F_z,
                                           int nx, int ny, int nz, int n_ghost,
                                           Real dx, Real dy, Real dz, Real dt, int n_fields);


Real CTU_Algorithm_3D_CUDA(Real *host_conserved0, Real *host_conserved1, int nx, int ny, int nz, int x_off, int y_off, int z_off, int n_ghost, Real dx, Real dy, Real dz, Real xbound, Real ybound, Real zbound, Real dt, int n_fields, Real dens_floor, Real temp_floor)
{
  //Here, *host_conserved contains the entire
  //set of conserved variables on the grid
  //concatenated into a 1-d array
  //host_conserved0 contains the values at time n,
  //host_conserved1 contains the values at time n+1

  // dimensions of subgrid blocks
  int nx_s, ny_s, nz_s;
  // x, y, and z offsets for subgrid blocks
  int x_off_s, y_off_s, z_off_s;
  // total number of subgrid blocks needed
  int block_tot;
  // number of subgrid blocks needed in each direction
  int block1_tot, block2_tot, block3_tot;
  // modulus of number of cells after block subdivision in each direction
  int remainder1, remainder2, remainder3;

  // counter for which block we're on
  int block = 0;

  // calculate the dimensions for the subgrid blocks
  sub_dimensions_3D(nx, ny, nz, n_ghost, &nx_s, &ny_s, &nz_s, &block1_tot, &block2_tot, &block3_tot, &remainder1, &remainder2, &remainder3, n_fields);
  block_tot = block1_tot*block2_tot*block3_tot;

  // number of cells in one subgrid block
  int BLOCK_VOL = nx_s*ny_s*nz_s;

  // dimensions for the 1D GPU grid
  int  ngrid = (BLOCK_VOL + TPB - 1) / TPB;

  //number of blocks per 1D grid
  dim3 dim1dGrid(ngrid, 1, 1);

  //number of threads per 1D block
  dim3 dim1dBlock(TPB, 1, 1);

  // Set up pointers for the location to copy from and to
  Real *tmp1;
  Real *tmp2;

  // allocate buffer to copy conserved variable blocks to/from
  Real *buffer;
  if (block_tot > 1) {
    if ( NULL == ( buffer = (Real *) malloc(n_fields*BLOCK_VOL*sizeof(Real)) ) ) {
      printf("Failed to allocate CPU buffer.\n");
    }
    tmp1 = buffer;
    tmp2 = buffer;
  }
  else {
    tmp1 = host_conserved0;
    tmp2 = host_conserved1;
  }

  // allocate an array on the CPU to hold max_dti returned from each thread block
  Real max_dti = 0;
  Real *host_dti_array;
  host_dti_array = (Real *) malloc(ngrid*sizeof(Real));
  #ifdef COOLING_GPU
  Real min_dt = 1e10;
  Real *host_dt_array;
  host_dt_array = (Real *) malloc(ngrid*sizeof(Real));
  #endif

  // allocate GPU arrays
  // conserved variables
  Real *dev_conserved;
  // input states and associated interface fluxes (Q* and F* from Stone, 2008)
  Real *Q_Lx, *Q_Rx, *Q_Ly, *Q_Ry, *Q_Lz, *Q_Rz, *F_x, *F_y, *F_z;
  // arrays to hold the eta values for the H correction
  Real *eta_x, *eta_y, *eta_z, *etah_x, *etah_y, *etah_z;
  // array of inverse timesteps for dt calculation
  Real *dev_dti_array;
  #ifdef COOLING_GPU
  // array of timesteps for dt calculation (cooling restriction)
  Real *dev_dt_array;
  #endif

  // allocate memory on the GPU
  CudaSafeCall( cudaMalloc((void**)&dev_conserved, n_fields*BLOCK_VOL*sizeof(Real)) );
  CudaSafeCall( cudaMalloc((void**)&Q_Lx,  n_fields*BLOCK_VOL*sizeof(Real)) );
  CudaSafeCall( cudaMalloc((void**)&Q_Rx,  n_fields*BLOCK_VOL*sizeof(Real)) );
  CudaSafeCall( cudaMalloc((void**)&Q_Ly,  n_fields*BLOCK_VOL*sizeof(Real)) );
  CudaSafeCall( cudaMalloc((void**)&Q_Ry,  n_fields*BLOCK_VOL*sizeof(Real)) );
  CudaSafeCall( cudaMalloc((void**)&Q_Lz,  n_fields*BLOCK_VOL*sizeof(Real)) );
  CudaSafeCall( cudaMalloc((void**)&Q_Rz,  n_fields*BLOCK_VOL*sizeof(Real)) );
  CudaSafeCall( cudaMalloc((void**)&F_x,   n_fields*BLOCK_VOL*sizeof(Real)) );
  CudaSafeCall( cudaMalloc((void**)&F_y,   n_fields*BLOCK_VOL*sizeof(Real)) );
  CudaSafeCall( cudaMalloc((void**)&F_z,   n_fields*BLOCK_VOL*sizeof(Real)) );
  CudaSafeCall( cudaMalloc((void**)&eta_x,  BLOCK_VOL*sizeof(Real)) );
  CudaSafeCall( cudaMalloc((void**)&eta_y,  BLOCK_VOL*sizeof(Real)) );
  CudaSafeCall( cudaMalloc((void**)&eta_z,  BLOCK_VOL*sizeof(Real)) );
  CudaSafeCall( cudaMalloc((void**)&etah_x, BLOCK_VOL*sizeof(Real)) );
  CudaSafeCall( cudaMalloc((void**)&etah_y, BLOCK_VOL*sizeof(Real)) );
  CudaSafeCall( cudaMalloc((void**)&etah_z, BLOCK_VOL*sizeof(Real)) );
  CudaSafeCall( cudaMalloc((void**)&dev_dti_array, ngrid*sizeof(Real)) );
  #ifdef COOLING_GPU
  CudaSafeCall( cudaMalloc((void**)&dev_dt_array, ngrid*sizeof(Real)) );
  #endif

  // START LOOP OVER SUBGRID BLOCKS
  while (block < block_tot) {

    // copy the conserved variable block to the buffer
    host_copy_block_3D(nx, ny, nz, nx_s, ny_s, nz_s, n_ghost, block, block1_tot, block2_tot, block3_tot, remainder1, remainder2, remainder3, BLOCK_VOL, host_conserved0, buffer, n_fields);

    get_offsets_3D(nx_s, ny_s, nz_s, n_ghost, x_off, y_off, z_off, block, block1_tot, block2_tot, block3_tot, remainder1, remainder2, remainder3, &x_off_s, &y_off_s, &z_off_s);

    // copy the conserved variables onto the GPU
    CudaSafeCall( cudaMemcpy(dev_conserved, tmp1, n_fields*BLOCK_VOL*sizeof(Real), cudaMemcpyHostToDevice) );


    // Step 1: Do the reconstruction
    #ifdef PCM
    PCM_Reconstruction_3D<<<dim1dGrid,dim1dBlock>>>(dev_conserved, Q_Lx, Q_Rx, Q_Ly, Q_Ry, Q_Lz, Q_Rz, nx_s, ny_s, nz_s, n_ghost, gama, n_fields);
    #endif //PCM
    #ifdef PLMP
    PLMP_cuda<<<dim1dGrid,dim1dBlock>>>(dev_conserved, Q_Lx, Q_Rx, nx_s, ny_s, nz_s, n_ghost, dx, dt, gama, 0, n_fields);
    PLMP_cuda<<<dim1dGrid,dim1dBlock>>>(dev_conserved, Q_Ly, Q_Ry, nx_s, ny_s, nz_s, n_ghost, dy, dt, gama, 1, n_fields);
    PLMP_cuda<<<dim1dGrid,dim1dBlock>>>(dev_conserved, Q_Lz, Q_Rz, nx_s, ny_s, nz_s, n_ghost, dz, dt, gama, 2, n_fields);
    #endif //PLMP
    #ifdef PLMC
    PLMC_cuda<<<dim1dGrid,dim1dBlock>>>(dev_conserved, Q_Lx, Q_Rx, nx_s, ny_s, nz_s, n_ghost, dx, dt, gama, 0, n_fields);
    PLMC_cuda<<<dim1dGrid,dim1dBlock>>>(dev_conserved, Q_Ly, Q_Ry, nx_s, ny_s, nz_s, n_ghost, dy, dt, gama, 1, n_fields);
    PLMC_cuda<<<dim1dGrid,dim1dBlock>>>(dev_conserved, Q_Lz, Q_Rz, nx_s, ny_s, nz_s, n_ghost, dz, dt, gama, 2, n_fields);
    #endif //PLMC
    #ifdef PPMP
    PPMP_cuda<<<dim1dGrid,dim1dBlock>>>(dev_conserved, Q_Lx, Q_Rx, nx_s, ny_s, nz_s, n_ghost, dx, dt, gama, 0, n_fields);
    PPMP_cuda<<<dim1dGrid,dim1dBlock>>>(dev_conserved, Q_Ly, Q_Ry, nx_s, ny_s, nz_s, n_ghost, dy, dt, gama, 1, n_fields);
    PPMP_cuda<<<dim1dGrid,dim1dBlock>>>(dev_conserved, Q_Lz, Q_Rz, nx_s, ny_s, nz_s, n_ghost, dz, dt, gama, 2, n_fields);
    #endif //PPMP
    #ifdef PPMC
    PPMC_cuda<<<dim1dGrid,dim1dBlock>>>(dev_conserved, Q_Lx, Q_Rx, nx_s, ny_s, nz_s, n_ghost, dx, dt, gama, 0, n_fields);
    PPMC_cuda<<<dim1dGrid,dim1dBlock>>>(dev_conserved, Q_Ly, Q_Ry, nx_s, ny_s, nz_s, n_ghost, dy, dt, gama, 1, n_fields);
    PPMC_cuda<<<dim1dGrid,dim1dBlock>>>(dev_conserved, Q_Lz, Q_Rz, nx_s, ny_s, nz_s, n_ghost, dz, dt, gama, 2, n_fields);
    #endif //PPMC
    CudaCheckError();


    #ifdef H_CORRECTION
    #ifndef CTU
    calc_eta_x_3D<<<dim1dGrid,dim1dBlock>>>(Q_Lx, Q_Rx, eta_x, nx_s, ny_s, nz_s, n_ghost, gama);
    calc_eta_y_3D<<<dim1dGrid,dim1dBlock>>>(Q_Ly, Q_Ry, eta_y, nx_s, ny_s, nz_s, n_ghost, gama);
    calc_eta_z_3D<<<dim1dGrid,dim1dBlock>>>(Q_Lz, Q_Rz, eta_z, nx_s, ny_s, nz_s, n_ghost, gama);
    CudaCheckError();
    // and etah values for each interface
    calc_etah_x_3D<<<dim1dGrid,dim1dBlock>>>(eta_x, eta_y, eta_z, etah_x, nx_s, ny_s, nz_s, n_ghost);
    calc_etah_y_3D<<<dim1dGrid,dim1dBlock>>>(eta_x, eta_y, eta_z, etah_y, nx_s, ny_s, nz_s, n_ghost);
    calc_etah_z_3D<<<dim1dGrid,dim1dBlock>>>(eta_x, eta_y, eta_z, etah_z, nx_s, ny_s, nz_s, n_ghost);
    CudaCheckError();
    #endif // NO CTU
    #endif // H_CORRECTION


    // Step 2: Calculate the fluxes
    #ifdef EXACT
    Calculate_Exact_Fluxes_CUDA<<<dim1dGrid,dim1dBlock>>>(Q_Lx, Q_Rx, F_x, nx_s, ny_s, nz_s, n_ghost, gama, 0, n_fields);
    Calculate_Exact_Fluxes_CUDA<<<dim1dGrid,dim1dBlock>>>(Q_Ly, Q_Ry, F_y, nx_s, ny_s, nz_s, n_ghost, gama, 1, n_fields);
    Calculate_Exact_Fluxes_CUDA<<<dim1dGrid,dim1dBlock>>>(Q_Lz, Q_Rz, F_z, nx_s, ny_s, nz_s, n_ghost, gama, 2, n_fields);
    #endif //EXACT
    #ifdef ROE
    Calculate_Roe_Fluxes_CUDA<<<dim1dGrid,dim1dBlock>>>(Q_Lx, Q_Rx, F_x, nx_s, ny_s, nz_s, n_ghost, gama, etah_x, 0, n_fields);
    Calculate_Roe_Fluxes_CUDA<<<dim1dGrid,dim1dBlock>>>(Q_Ly, Q_Ry, F_y, nx_s, ny_s, nz_s, n_ghost, gama, etah_y, 1, n_fields);
    Calculate_Roe_Fluxes_CUDA<<<dim1dGrid,dim1dBlock>>>(Q_Lz, Q_Rz, F_z, nx_s, ny_s, nz_s, n_ghost, gama, etah_z, 2, n_fields);
    #endif //ROE
    #ifdef HLLC
    Calculate_HLLC_Fluxes_CUDA<<<dim1dGrid,dim1dBlock>>>(Q_Lx, Q_Rx, F_x, nx_s, ny_s, nz_s, n_ghost, gama, etah_x, 0, n_fields);
    Calculate_HLLC_Fluxes_CUDA<<<dim1dGrid,dim1dBlock>>>(Q_Ly, Q_Ry, F_y, nx_s, ny_s, nz_s, n_ghost, gama, etah_y, 1, n_fields);
    Calculate_HLLC_Fluxes_CUDA<<<dim1dGrid,dim1dBlock>>>(Q_Lz, Q_Rz, F_z, nx_s, ny_s, nz_s, n_ghost, gama, etah_z, 2, n_fields);
    #endif //HLLC
    CudaCheckError();


    #ifdef CTU
    // Step 3: Evolve the interface states
    Evolve_Interface_States_3D<<<dim1dGrid,dim1dBlock>>>(dev_conserved, Q_Lx, Q_Rx, F_x, Q_Ly, Q_Ry, F_y, Q_Lz, Q_Rz, F_z, nx_s, ny_s, nz_s, n_ghost, dx, dy, dz, dt, n_fields);
    CudaCheckError();



    #ifdef H_CORRECTION
    // Step 3.5: Calculate eta values for H correction
    calc_eta_x_3D<<<dim1dGrid,dim1dBlock>>>(Q_Lx, Q_Rx, eta_x, nx_s, ny_s, nz_s, n_ghost, gama);
    calc_eta_y_3D<<<dim1dGrid,dim1dBlock>>>(Q_Ly, Q_Ry, eta_y, nx_s, ny_s, nz_s, n_ghost, gama);
    calc_eta_z_3D<<<dim1dGrid,dim1dBlock>>>(Q_Lz, Q_Rz, eta_z, nx_s, ny_s, nz_s, n_ghost, gama);
    CudaCheckError();
    // and etah values for each interface
    calc_etah_x_3D<<<dim1dGrid,dim1dBlock>>>(eta_x, eta_y, eta_z, etah_x, nx_s, ny_s, nz_s, n_ghost);
    calc_etah_y_3D<<<dim1dGrid,dim1dBlock>>>(eta_x, eta_y, eta_z, etah_y, nx_s, ny_s, nz_s, n_ghost);
    calc_etah_z_3D<<<dim1dGrid,dim1dBlock>>>(eta_x, eta_y, eta_z, etah_z, nx_s, ny_s, nz_s, n_ghost);
    CudaCheckError();
    #endif //H_CORRECTION


    // Step 4: Calculate the fluxes again
    #ifdef EXACT
    Calculate_Exact_Fluxes_CUDA<<<dim1dGrid,dim1dBlock>>>(Q_Lx, Q_Rx, F_x, nx_s, ny_s, nz_s, n_ghost, gama, 0, n_fields);
    Calculate_Exact_Fluxes_CUDA<<<dim1dGrid,dim1dBlock>>>(Q_Ly, Q_Ry, F_y, nx_s, ny_s, nz_s, n_ghost, gama, 1, n_fields);
    Calculate_Exact_Fluxes_CUDA<<<dim1dGrid,dim1dBlock>>>(Q_Lz, Q_Rz, F_z, nx_s, ny_s, nz_s, n_ghost, gama, 2, n_fields);
    #endif //EXACT
    #ifdef ROE
    Calculate_Roe_Fluxes_CUDA<<<dim1dGrid,dim1dBlock>>>(Q_Lx, Q_Rx, F_x, nx_s, ny_s, nz_s, n_ghost, gama, etah_x, 0, n_fields);
    Calculate_Roe_Fluxes_CUDA<<<dim1dGrid,dim1dBlock>>>(Q_Ly, Q_Ry, F_y, nx_s, ny_s, nz_s, n_ghost, gama, etah_y, 1, n_fields);
    Calculate_Roe_Fluxes_CUDA<<<dim1dGrid,dim1dBlock>>>(Q_Lz, Q_Rz, F_z, nx_s, ny_s, nz_s, n_ghost, gama, etah_z, 2, n_fields);
    #endif //ROE
    #ifdef HLLC
    Calculate_HLLC_Fluxes_CUDA<<<dim1dGrid,dim1dBlock>>>(Q_Lx, Q_Rx, F_x, nx_s, ny_s, nz_s, n_ghost, gama, etah_x, 0, n_fields);
    Calculate_HLLC_Fluxes_CUDA<<<dim1dGrid,dim1dBlock>>>(Q_Ly, Q_Ry, F_y, nx_s, ny_s, nz_s, n_ghost, gama, etah_y, 1, n_fields);
    Calculate_HLLC_Fluxes_CUDA<<<dim1dGrid,dim1dBlock>>>(Q_Lz, Q_Rz, F_z, nx_s, ny_s, nz_s, n_ghost, gama, etah_z, 2, n_fields);
    #endif //HLLC
    CudaCheckError();
    #endif //CTU

    // Step 5: Update the conserved variable array
    Update_Conserved_Variables_3D<<<dim1dGrid,dim1dBlock>>>(dev_conserved, F_x, F_y, F_z, nx_s, ny_s, nz_s, x_off, y_off, z_off, n_ghost, dx, dy, dz, xbound, ybound, zbound, dt, gama, n_fields, dens_floor);
    CudaCheckError();


    // Synchronize the total and internal energies
    #ifndef GRAVITY_CPU
    #ifdef DE
    Sync_Energies_3D<<<dim1dGrid,dim1dBlock>>>(dev_conserved, nx_s, ny_s, nz_s, n_ghost, gama, n_fields);
    CudaCheckError();

    #ifdef TEMPERATURE_FLOOR
    Apply_Temperature_Floor<<<dim1dGrid,dim1dBlock>>>(dev_conserved, nx_s, ny_s, nz_s, n_ghost, n_fields, temp_floor );
    CudaCheckError();
    #endif //TEMPERATURE_FLOOR
    #endif //DE
    #endif //GRAVITY_CPU


    // Apply cooling
    #ifdef COOLING_GPU
    cooling_kernel<<<dim1dGrid,dim1dBlock>>>(dev_conserved, nx_s, ny_s, nz_s, n_ghost, n_fields, dt, gama, dev_dt_array);
    CudaCheckError();
    #endif


    // Step 6: Calculate the next timestep
    Calc_dt_3D<<<dim1dGrid,dim1dBlock>>>(dev_conserved, nx_s, ny_s, nz_s, n_ghost, dx, dy, dz, dev_dti_array, gama);
    CudaCheckError();



    // copy the updated conserved variable array back to the CPU
    CudaSafeCall( cudaMemcpy(tmp2, dev_conserved, n_fields*BLOCK_VOL*sizeof(Real), cudaMemcpyDeviceToHost) );
    CudaCheckError();

    // copy the updated conserved variable array from the buffer into the host_conserved array on the CPU
    host_return_block_3D(nx, ny, nz, nx_s, ny_s, nz_s, n_ghost, block, block1_tot, block2_tot, block3_tot, remainder1, remainder2, remainder3, BLOCK_VOL, host_conserved1, buffer, n_fields);

    // copy the dti array onto the CPU
    CudaSafeCall( cudaMemcpy(host_dti_array, dev_dti_array, ngrid*sizeof(Real), cudaMemcpyDeviceToHost) );
    // iterate through to find the maximum inverse dt for this subgrid block
    for (int i=0; i<ngrid; i++) {
      max_dti = fmax(max_dti, host_dti_array[i]);
    }
    #ifdef COOLING_GPU
    // copy the dt array from cooling onto the CPU
    CudaSafeCall( cudaMemcpy(host_dt_array, dev_dt_array, ngrid*sizeof(Real), cudaMemcpyDeviceToHost) );
    // find maximum inverse timestep from cooling time
    for (int i=0; i<ngrid; i++) {
      min_dt = fmin(min_dt, host_dt_array[i]);
    }
    if (min_dt < C_cfl/max_dti) {
      max_dti = C_cfl/min_dt;
    }
    #endif

    // add one to the counter
    block++;

  }


  // free CPU memory
  free(host_dti_array);
  if (block_tot > 1) free(buffer);
  #ifdef COOLING_GPU
  free(host_dt_array);
  #endif

  // free the GPU memory
  cudaFree(dev_conserved);
  cudaFree(Q_Lx);
  cudaFree(Q_Rx);
  cudaFree(Q_Ly);
  cudaFree(Q_Ry);
  cudaFree(Q_Lz);
  cudaFree(Q_Rz);
  cudaFree(F_x);
  cudaFree(F_y);
  cudaFree(F_z);
  cudaFree(eta_x);
  cudaFree(eta_y);
  cudaFree(eta_z);
  cudaFree(etah_x);
  cudaFree(etah_y);
  cudaFree(etah_z);
  cudaFree(dev_dti_array);
  #ifdef COOLING_GPU
  cudaFree(dev_dt_array);
  #endif

  // return the maximum inverse timestep
  return max_dti;

}


__global__ void Evolve_Interface_States_3D(Real *dev_conserved, Real *dev_Q_Lx, Real *dev_Q_Rx, Real *dev_F_x,
                                           Real *dev_Q_Ly, Real *dev_Q_Ry, Real *dev_F_y,
                                           Real *dev_Q_Lz, Real *dev_Q_Rz, Real *dev_F_z,
                                           int nx, int ny, int nz, int n_ghost, Real dx, Real dy, Real dz, Real dt, int n_fields)
{
  Real dtodx = dt/dx;
  Real dtody = dt/dy;
  Real dtodz = dt/dz;
  int n_cells = nx*ny*nz;

  // get a thread ID
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  int zid = tid / (nx*ny);
  int yid = (tid - zid*nx*ny) / nx;
  int xid = tid - zid*nx*ny - yid*nx;
  int id = xid + yid*nx + zid*nx*ny;

  if (xid > n_ghost-3 && xid < nx-n_ghost+1 && yid > n_ghost-2 && yid < ny-n_ghost+1 && zid > n_ghost-2 && zid < nz-n_ghost+1)
  {
    // set the new x interface states
    // left
    int ipo = xid+1 + yid*nx + zid*nx*ny;
    int jmo = xid + (yid-1)*nx + zid*nx*ny;
    int kmo = xid + yid*nx + (zid-1)*nx*ny;
    int ipojmo = xid+1 + (yid-1)*nx + zid*nx*ny;
    int ipokmo = xid+1 + yid*nx + (zid-1)*nx*ny;
    dev_Q_Lx[            id] += 0.5*dtody*(dev_F_y[            jmo] - dev_F_y[            id])
                              + 0.5*dtodz*(dev_F_z[            kmo] - dev_F_z[            id]);
    dev_Q_Lx[  n_cells + id] += 0.5*dtody*(dev_F_y[  n_cells + jmo] - dev_F_y[  n_cells + id])
                              + 0.5*dtodz*(dev_F_z[  n_cells + kmo] - dev_F_z[  n_cells + id]);
    dev_Q_Lx[2*n_cells + id] += 0.5*dtody*(dev_F_y[2*n_cells + jmo] - dev_F_y[2*n_cells + id])
                              + 0.5*dtodz*(dev_F_z[2*n_cells + kmo] - dev_F_z[2*n_cells + id]);
    dev_Q_Lx[3*n_cells + id] += 0.5*dtody*(dev_F_y[3*n_cells + jmo] - dev_F_y[3*n_cells + id])
                              + 0.5*dtodz*(dev_F_z[3*n_cells + kmo] - dev_F_z[3*n_cells + id]);
    dev_Q_Lx[4*n_cells + id] += 0.5*dtody*(dev_F_y[4*n_cells + jmo] - dev_F_y[4*n_cells + id])
                              + 0.5*dtodz*(dev_F_z[4*n_cells + kmo] - dev_F_z[4*n_cells + id]);
    #ifdef SCALAR
    for (int i=0; i<NSCALARS; i++) {
      dev_Q_Lx[(5+i)*n_cells + id] += 0.5*dtody*(dev_F_y[(5+i)*n_cells + jmo] - dev_F_y[(5+i)*n_cells + id])
                                + 0.5*dtodz*(dev_F_z[(5+i)*n_cells + kmo] - dev_F_z[(5+i)*n_cells + id]);
    }
    #endif
    #ifdef DE
    dev_Q_Lx[(n_fields-1)*n_cells + id] += 0.5*dtody*(dev_F_y[(n_fields-1)*n_cells + jmo] - dev_F_y[(n_fields-1)*n_cells + id])
                              + 0.5*dtodz*(dev_F_z[(n_fields-1)*n_cells + kmo] - dev_F_z[(n_fields-1)*n_cells + id]);
    #endif

    // right
    dev_Q_Rx[            id] += 0.5*dtody*(dev_F_y[            ipojmo] - dev_F_y[            ipo])
                              + 0.5*dtodz*(dev_F_z[            ipokmo] - dev_F_z[            ipo]);
    dev_Q_Rx[  n_cells + id] += 0.5*dtody*(dev_F_y[  n_cells + ipojmo] - dev_F_y[  n_cells + ipo])
                              + 0.5*dtodz*(dev_F_z[  n_cells + ipokmo] - dev_F_z[  n_cells + ipo]);
    dev_Q_Rx[2*n_cells + id] += 0.5*dtody*(dev_F_y[2*n_cells + ipojmo] - dev_F_y[2*n_cells + ipo])
                              + 0.5*dtodz*(dev_F_z[2*n_cells + ipokmo] - dev_F_z[2*n_cells + ipo]);
    dev_Q_Rx[3*n_cells + id] += 0.5*dtody*(dev_F_y[3*n_cells + ipojmo] - dev_F_y[3*n_cells + ipo])
                              + 0.5*dtodz*(dev_F_z[3*n_cells + ipokmo] - dev_F_z[3*n_cells + ipo]);
    dev_Q_Rx[4*n_cells + id] += 0.5*dtody*(dev_F_y[4*n_cells + ipojmo] - dev_F_y[4*n_cells + ipo])
                              + 0.5*dtodz*(dev_F_z[4*n_cells + ipokmo] - dev_F_z[4*n_cells + ipo]);
    #ifdef SCALAR
    for (int i=0; i<NSCALARS; i++) {
      dev_Q_Rx[(5+i)*n_cells + id] += 0.5*dtody*(dev_F_y[(5+i)*n_cells + ipojmo] - dev_F_y[(5+i)*n_cells + ipo])
                                + 0.5*dtodz*(dev_F_z[(5+i)*n_cells + ipokmo] - dev_F_z[(5+i)*n_cells + ipo]);
    }
    #endif
    #ifdef DE
    dev_Q_Rx[(n_fields-1)*n_cells + id] += 0.5*dtody*(dev_F_y[(n_fields-1)*n_cells + ipojmo] - dev_F_y[(n_fields-1)*n_cells + ipo])
                              + 0.5*dtodz*(dev_F_z[(n_fields-1)*n_cells + ipokmo] - dev_F_z[(n_fields-1)*n_cells + ipo]);
    #endif
  }
  if (yid > n_ghost-3 && yid < ny-n_ghost+1 && xid > n_ghost-2 && xid < nx-n_ghost+1 && zid > n_ghost-2 && zid < nz-n_ghost+1)
  {
    // set the new y interface states
    // left
    int jpo = xid + (yid+1)*nx + zid*nx*ny;
    int imo = xid-1 + yid*nx + zid*nx*ny;
    int kmo = xid + yid*nx + (zid-1)*nx*ny;
    int jpoimo = xid-1 + (yid+1)*nx + zid*nx*ny;
    int jpokmo = xid + (yid+1)*nx + (zid-1)*nx*ny;
    dev_Q_Ly[            id] += 0.5*dtodz*(dev_F_z[            kmo] - dev_F_z[            id])
                              + 0.5*dtodx*(dev_F_x[            imo] - dev_F_x[            id]);
    dev_Q_Ly[  n_cells + id] += 0.5*dtodz*(dev_F_z[  n_cells + kmo] - dev_F_z[  n_cells + id])
                              + 0.5*dtodx*(dev_F_x[  n_cells + imo] - dev_F_x[  n_cells + id]);
    dev_Q_Ly[2*n_cells + id] += 0.5*dtodz*(dev_F_z[2*n_cells + kmo] - dev_F_z[2*n_cells + id])
                              + 0.5*dtodx*(dev_F_x[2*n_cells + imo] - dev_F_x[2*n_cells + id]);
    dev_Q_Ly[3*n_cells + id] += 0.5*dtodz*(dev_F_z[3*n_cells + kmo] - dev_F_z[3*n_cells + id])
                              + 0.5*dtodx*(dev_F_x[3*n_cells + imo] - dev_F_x[3*n_cells + id]);
    dev_Q_Ly[4*n_cells + id] += 0.5*dtodz*(dev_F_z[4*n_cells + kmo] - dev_F_z[4*n_cells + id])
                              + 0.5*dtodx*(dev_F_x[4*n_cells + imo] - dev_F_x[4*n_cells + id]);
    #ifdef SCALAR
    for (int i=0; i<NSCALARS; i++) {
      dev_Q_Ly[(5+i)*n_cells + id] += 0.5*dtodz*(dev_F_z[(5+i)*n_cells + kmo] - dev_F_z[(5+i)*n_cells + id])
                                + 0.5*dtodx*(dev_F_x[(5+i)*n_cells + imo] - dev_F_x[(5+i)*n_cells + id]);
    }
    #endif
    #ifdef DE
    dev_Q_Ly[(n_fields-1)*n_cells + id] += 0.5*dtodz*(dev_F_z[(n_fields-1)*n_cells + kmo] - dev_F_z[(n_fields-1)*n_cells + id])
                              + 0.5*dtodx*(dev_F_x[(n_fields-1)*n_cells + imo] - dev_F_x[(n_fields-1)*n_cells + id]);
    #endif

    // right
    dev_Q_Ry[            id] += 0.5*dtodz*(dev_F_z[            jpokmo] - dev_F_z[            jpo])
                              + 0.5*dtodx*(dev_F_x[            jpoimo] - dev_F_x[            jpo]);
    dev_Q_Ry[  n_cells + id] += 0.5*dtodz*(dev_F_z[  n_cells + jpokmo] - dev_F_z[  n_cells + jpo])
                              + 0.5*dtodx*(dev_F_x[  n_cells + jpoimo] - dev_F_x[  n_cells + jpo]);
    dev_Q_Ry[2*n_cells + id] += 0.5*dtodz*(dev_F_z[2*n_cells + jpokmo] - dev_F_z[2*n_cells + jpo])
                              + 0.5*dtodx*(dev_F_x[2*n_cells + jpoimo] - dev_F_x[2*n_cells + jpo]);
    dev_Q_Ry[3*n_cells + id] += 0.5*dtodz*(dev_F_z[3*n_cells + jpokmo] - dev_F_z[3*n_cells + jpo])
                              + 0.5*dtodx*(dev_F_x[3*n_cells + jpoimo] - dev_F_x[3*n_cells + jpo]);
    dev_Q_Ry[4*n_cells + id] += 0.5*dtodz*(dev_F_z[4*n_cells + jpokmo] - dev_F_z[4*n_cells + jpo])
                              + 0.5*dtodx*(dev_F_x[4*n_cells + jpoimo] - dev_F_x[4*n_cells + jpo]);
    #ifdef SCALAR
    for (int i=0; i<NSCALARS; i++) {
      dev_Q_Ry[(5+i)*n_cells + id] += 0.5*dtodz*(dev_F_z[(5+i)*n_cells + jpokmo] - dev_F_z[(5+i)*n_cells + jpo])
                                + 0.5*dtodx*(dev_F_x[(5+i)*n_cells + jpoimo] - dev_F_x[(5+i)*n_cells + jpo]);
    }
    #endif
    #ifdef DE
    dev_Q_Ry[(n_fields-1)*n_cells + id] += 0.5*dtodz*(dev_F_z[(n_fields-1)*n_cells + jpokmo] - dev_F_z[(n_fields-1)*n_cells + jpo])
                              + 0.5*dtodx*(dev_F_x[(n_fields-1)*n_cells + jpoimo] - dev_F_x[(n_fields-1)*n_cells + jpo]);
    #endif
  }
  if (zid > n_ghost-3 && zid < nz-n_ghost+1 && xid > n_ghost-2 && xid < nx-n_ghost+1 && yid > n_ghost-2 && yid < ny-n_ghost+1)
  {
    // set the new z interface states
    // left
    int kpo = xid + yid*nx + (zid+1)*nx*ny;
    int imo = xid-1 + yid*nx + zid*nx*ny;
    int jmo = xid + (yid-1)*nx + zid*nx*ny;
    int kpoimo = xid-1 + yid*nx + (zid+1)*nx*ny;
    int kpojmo = xid + (yid-1)*nx + (zid+1)*nx*ny;
    dev_Q_Lz[            id] += 0.5*dtodx*(dev_F_x[            imo] - dev_F_x[            id])
                              + 0.5*dtody*(dev_F_y[            jmo] - dev_F_y[            id]);
    dev_Q_Lz[  n_cells + id] += 0.5*dtodx*(dev_F_x[  n_cells + imo] - dev_F_x[  n_cells + id])
                              + 0.5*dtody*(dev_F_y[  n_cells + jmo] - dev_F_y[  n_cells + id]);
    dev_Q_Lz[2*n_cells + id] += 0.5*dtodx*(dev_F_x[2*n_cells + imo] - dev_F_x[2*n_cells + id])
                              + 0.5*dtody*(dev_F_y[2*n_cells + jmo] - dev_F_y[2*n_cells + id]);
    dev_Q_Lz[3*n_cells + id] += 0.5*dtodx*(dev_F_x[3*n_cells + imo] - dev_F_x[3*n_cells + id])
                              + 0.5*dtody*(dev_F_y[3*n_cells + jmo] - dev_F_y[3*n_cells + id]);
    dev_Q_Lz[4*n_cells + id] += 0.5*dtodx*(dev_F_x[4*n_cells + imo] - dev_F_x[4*n_cells + id])
                              + 0.5*dtody*(dev_F_y[4*n_cells + jmo] - dev_F_y[4*n_cells + id]);
    #ifdef SCALAR
    for (int i=0; i<NSCALARS; i++) {
      dev_Q_Lz[(5+i)*n_cells + id] += 0.5*dtodx*(dev_F_x[(5+i)*n_cells + imo] - dev_F_x[(5+i)*n_cells + id])
                                + 0.5*dtody*(dev_F_y[(5+i)*n_cells + jmo] - dev_F_y[(5+i)*n_cells + id]);
    }
    #endif
    #ifdef DE
    dev_Q_Lz[(n_fields-1)*n_cells + id] += 0.5*dtodx*(dev_F_x[(n_fields-1)*n_cells + imo] - dev_F_x[(n_fields-1)*n_cells + id])
                              + 0.5*dtody*(dev_F_y[(n_fields-1)*n_cells + jmo] - dev_F_y[(n_fields-1)*n_cells + id]);
    #endif
    // right
    dev_Q_Rz[            id] += 0.5*dtodx*(dev_F_x[            kpoimo] - dev_F_x[            kpo])
                              + 0.5*dtody*(dev_F_y[            kpojmo] - dev_F_y[            kpo]);
    dev_Q_Rz[  n_cells + id] += 0.5*dtodx*(dev_F_x[  n_cells + kpoimo] - dev_F_x[  n_cells + kpo])
                              + 0.5*dtody*(dev_F_y[  n_cells + kpojmo] - dev_F_y[  n_cells + kpo]);
    dev_Q_Rz[2*n_cells + id] += 0.5*dtodx*(dev_F_x[2*n_cells + kpoimo] - dev_F_x[2*n_cells + kpo])
                              + 0.5*dtody*(dev_F_y[2*n_cells + kpojmo] - dev_F_y[2*n_cells + kpo]);
    dev_Q_Rz[3*n_cells + id] += 0.5*dtodx*(dev_F_x[3*n_cells + kpoimo] - dev_F_x[3*n_cells + kpo])
                              + 0.5*dtody*(dev_F_y[3*n_cells + kpojmo] - dev_F_y[3*n_cells + kpo]);
    dev_Q_Rz[4*n_cells + id] += 0.5*dtodx*(dev_F_x[4*n_cells + kpoimo] - dev_F_x[4*n_cells + kpo])
                              + 0.5*dtody*(dev_F_y[4*n_cells + kpojmo] - dev_F_y[4*n_cells + kpo]);
    #ifdef SCALAR
    for (int i=0; i<NSCALARS; i++) {
      dev_Q_Rz[(5+i)*n_cells + id] += 0.5*dtodx*(dev_F_x[(5+i)*n_cells + kpoimo] - dev_F_x[(5+i)*n_cells + kpo])
                                + 0.5*dtody*(dev_F_y[(5+i)*n_cells + kpojmo] - dev_F_y[(5+i)*n_cells + kpo]);
    }
    #endif
    #ifdef DE
    dev_Q_Rz[(n_fields-1)*n_cells + id] += 0.5*dtodx*(dev_F_x[(n_fields-1)*n_cells + kpoimo] - dev_F_x[(n_fields-1)*n_cells + kpo])
                              + 0.5*dtody*(dev_F_y[(n_fields-1)*n_cells + kpojmo] - dev_F_y[(n_fields-1)*n_cells + kpo]);
    #endif
  }

}



#endif //CUDA
