
#include <hip/hip_runtime.h>
#ifdef PARTICLES
#ifdef PARTICLES_CUDA

#include"density_CIC_CUDA.h"
#include"../io.h"
#include<cuda.h>
#include"../global_cuda.h"


void Get_Density_CIC_CUDA( Particles_3D &Parts ){

  part_int_t n_local = Parts.n_local;

  Real *mass_d;
  Real *pos_x_d;
  Real *pos_y_d;
  Real *pos_z_d;
  Real *density_d;

  // allocate memory on the GPU
  CudaSafeCall( cudaMalloc((void**)&mass_d,  n_local*sizeof(Real)) );
  CudaSafeCall( cudaMalloc((void**)&pos_x_d, n_local*sizeof(Real)) );
  CudaSafeCall( cudaMalloc((void**)&pos_y_d, n_local*sizeof(Real)) );
  CudaSafeCall( cudaMalloc((void**)&pos_z_d, n_local*sizeof(Real)) );
  CudaSafeCall( cudaMalloc((void**)&density_d, Parts.G.n_cells*sizeof(Real)) );
  //
  //
  //
  // free the GPU memory
  cudaFree(mass_d);
  cudaFree(pos_x_d);
  cudaFree(pos_y_d);
  cudaFree(pos_z_d);
  cudaFree(density_d);






}


#endif
#endif
