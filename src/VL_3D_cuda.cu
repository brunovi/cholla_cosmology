
#include <hip/hip_runtime.h>
/*! \file VL_3D_cuda.cu
 *  \brief Definitions of the cuda 3D VL algorithm functions. */

#ifdef CUDA
#ifdef VL

#include<stdio.h>
#include<stdlib.h>
#include<math.h>
#include<cuda.h>
#include"global.h"
#include"global_cuda.h"
#include"hydro_cuda.h"
#include"VL_3D_cuda.h"
#include"pcm_cuda.h"
#include"plmp_cuda.h"
#include"plmc_cuda.h"
#include"ppmp_cuda.h"
#include"ppmc_cuda.h"
#include"exact_cuda.h"
#include"roe_cuda.h"
#include"hllc_cuda.h"
#include"h_correction_3D_cuda.h"
#include"cooling_cuda.h"
#include"subgrid_routines_3D.h"
#include"io.h"

bool gpu_data_allocated;
int block;

// conserved variables
Real *dev_conserved, *dev_conserved_half;
// input states and associated interface fluxes (Q* and F* from Stone, 2008)
Real *Q_Lx, *Q_Rx, *Q_Ly, *Q_Ry, *Q_Lz, *Q_Rz, *F_x, *F_y, *F_z;
// arrays to hold the eta values for the H correction
Real *eta_x, *eta_y, *eta_z, *etah_x, *etah_y, *etah_z;
// array of inverse timesteps for dt calculation
Real *dev_dti_array;
#ifdef COOLING_GPU
// array of timesteps for dt calculation (cooling restriction)
Real *dev_dt_array;
#endif

__global__ void Update_Conserved_Variables_3D_half(Real *dev_conserved, Real *dev_conserved_half, Real *dev_F_x, Real *dev_F_y,  Real *dev_F_z, int nx, int ny, int nz, int n_ghost, Real dx, Real dy, Real dz, Real dt, Real gamma, int n_fields, Real dens_floor );



Real VL_Algorithm_3D_CUDA(Real *host_conserved0, Real *host_conserved1, int nx, int ny, int nz, int x_off, int y_off, int z_off, int n_ghost, Real dx, Real dy, Real dz, Real xbound, Real ybound, Real zbound, Real dt, int n_fields, Real dens_floor, Real temp_floor )
{

  //Here, *host_conserved contains the entire
  //set of conserved variables on the grid
  //concatenated into a 1-d array
  //host_conserved0 contains the values at time n,
  //host_conserved1 will contain the values at time n+1

  // dimensions of subgrid blocks
  int nx_s, ny_s, nz_s;
  // x, y, and z offsets for subgrid blocks
  int x_off_s, y_off_s, z_off_s;
  // total number of subgrid blocks needed
  int block_tot;
  // number of subgrid blocks needed in each direction
  int block1_tot, block2_tot, block3_tot;
  // modulus of number of cells after block subdivision in each direction
  int remainder1, remainder2, remainder3;

  // counter for which block we're on
  // int block = 0;
  block = 0;


  // calculate the dimensions for the subgrid blocks
  sub_dimensions_3D(nx, ny, nz, n_ghost, &nx_s, &ny_s, &nz_s, &block1_tot, &block2_tot, &block3_tot, &remainder1, &remainder2, &remainder3, n_fields);
  block_tot = block1_tot*block2_tot*block3_tot;

  // number of cells in one subgrid block
  int BLOCK_VOL = nx_s*ny_s*nz_s;

  // dimensions for the 1D GPU grid
  int  ngrid = (BLOCK_VOL + TPB - 1) / TPB;

  //number of blocks per 1D grid
  dim3 dim1dGrid(ngrid, 1, 1);

  //number of threads per 1D block
  dim3 dim1dBlock(TPB, 1, 1);

  // Set up pointers for the location to copy from and to
  Real *tmp1;
  Real *tmp2;

  // allocate buffer to copy conserved variable blocks to/from
  Real *buffer;
  if (block_tot > 1) {
    if ( NULL == ( buffer = (Real *) malloc(n_fields*BLOCK_VOL*sizeof(Real)) ) ) {
      printf("Failed to allocate CPU buffer.\n");
    }
    tmp1 = buffer;
    tmp2 = buffer;
  }
  else {
    tmp1 = host_conserved0;
    tmp2 = host_conserved1;
  }


  // allocate an array on the CPU to hold max_dti returned from each thread block
  Real max_dti = 0;
  Real *host_dti_array;
  host_dti_array = (Real *) malloc(ngrid*sizeof(Real));
  #ifdef COOLING_GPU
  Real min_dt = 1e10;
  Real *host_dt_array;
  host_dt_array = (Real *) malloc(ngrid*sizeof(Real));
  #endif

  // allocate GPU arrays
  // // conserved variables
  // Real *dev_conserved, *dev_conserved_half;
  // // input states and associated interface fluxes (Q* and F* from Stone, 2008)
  // Real *Q_Lx, *Q_Rx, *Q_Ly, *Q_Ry, *Q_Lz, *Q_Rz, *F_x, *F_y, *F_z;
  // // arrays to hold the eta values for the H correction
  // Real *eta_x, *eta_y, *eta_z, *etah_x, *etah_y, *etah_z;
  // // array of inverse timesteps for dt calculation
  // Real *dev_dti_array;
  // #ifdef COOLING_GPU
  // // array of timesteps for dt calculation (cooling restriction)
  // Real *dev_dt_array;
  // #endif

  if ( !gpu_data_allocated ){
    chprintf( " VL_3D: Allocating GPU memory \n");
    // allocate memory on the GPU
    CudaSafeCall( cudaMalloc((void**)&dev_conserved, n_fields*BLOCK_VOL*sizeof(Real)) );
    CudaSafeCall( cudaMalloc((void**)&dev_conserved_half, n_fields*BLOCK_VOL*sizeof(Real)) );
    CudaSafeCall( cudaMalloc((void**)&Q_Lx,  n_fields*BLOCK_VOL*sizeof(Real)) );
    CudaSafeCall( cudaMalloc((void**)&Q_Rx,  n_fields*BLOCK_VOL*sizeof(Real)) );
    CudaSafeCall( cudaMalloc((void**)&Q_Ly,  n_fields*BLOCK_VOL*sizeof(Real)) );
    CudaSafeCall( cudaMalloc((void**)&Q_Ry,  n_fields*BLOCK_VOL*sizeof(Real)) );
    CudaSafeCall( cudaMalloc((void**)&Q_Lz,  n_fields*BLOCK_VOL*sizeof(Real)) );
    CudaSafeCall( cudaMalloc((void**)&Q_Rz,  n_fields*BLOCK_VOL*sizeof(Real)) );
    CudaSafeCall( cudaMalloc((void**)&F_x,   n_fields*BLOCK_VOL*sizeof(Real)) );
    CudaSafeCall( cudaMalloc((void**)&F_y,   n_fields*BLOCK_VOL*sizeof(Real)) );
    CudaSafeCall( cudaMalloc((void**)&F_z,   n_fields*BLOCK_VOL*sizeof(Real)) );
    CudaSafeCall( cudaMalloc((void**)&eta_x,  BLOCK_VOL*sizeof(Real)) );
    CudaSafeCall( cudaMalloc((void**)&eta_y,  BLOCK_VOL*sizeof(Real)) );
    CudaSafeCall( cudaMalloc((void**)&eta_z,  BLOCK_VOL*sizeof(Real)) );
    CudaSafeCall( cudaMalloc((void**)&etah_x, BLOCK_VOL*sizeof(Real)) );
    CudaSafeCall( cudaMalloc((void**)&etah_y, BLOCK_VOL*sizeof(Real)) );
    CudaSafeCall( cudaMalloc((void**)&etah_z, BLOCK_VOL*sizeof(Real)) );
    CudaSafeCall( cudaMalloc((void**)&dev_dti_array, ngrid*sizeof(Real)) );
    #ifdef COOLING_GPU
    CudaSafeCall( cudaMalloc((void**)&dev_dt_array, ngrid*sizeof(Real)) );
    #endif
    gpu_data_allocated = true;
  }

  // START LOOP OVER SUBGRID BLOCKS
  while (block < block_tot) {

    // copy the conserved variable block to the buffer
    host_copy_block_3D(nx, ny, nz, nx_s, ny_s, nz_s, n_ghost, block, block1_tot, block2_tot, block3_tot, remainder1, remainder2, remainder3, BLOCK_VOL, host_conserved0, buffer, n_fields);

   // calculate the global x, y, and z offsets of this subgrid block
    get_offsets_3D(nx_s, ny_s, nz_s, n_ghost, x_off, y_off, z_off, block, block1_tot, block2_tot, block3_tot, remainder1, remainder2, remainder3, &x_off_s, &y_off_s, &z_off_s);

    // copy the conserved variables onto the GPU
    CudaSafeCall( cudaMemcpy(dev_conserved, tmp1, n_fields*BLOCK_VOL*sizeof(Real), cudaMemcpyHostToDevice) );


    // Step 1: Use PCM reconstruction to put primitive variables into interface arrays
    PCM_Reconstruction_3D<<<dim1dGrid,dim1dBlock>>>(dev_conserved, Q_Lx, Q_Rx, Q_Ly, Q_Ry, Q_Lz, Q_Rz, nx_s, ny_s, nz_s, n_ghost, gama, n_fields);
    CudaCheckError();


    // Step 2: Calculate first-order upwind fluxes
    #ifdef EXACT
    Calculate_Exact_Fluxes_CUDA<<<dim1dGrid,dim1dBlock>>>(Q_Lx, Q_Rx, F_x, nx_s, ny_s, nz_s, n_ghost, gama, 0, n_fields);
    Calculate_Exact_Fluxes_CUDA<<<dim1dGrid,dim1dBlock>>>(Q_Ly, Q_Ry, F_y, nx_s, ny_s, nz_s, n_ghost, gama, 1, n_fields);
    Calculate_Exact_Fluxes_CUDA<<<dim1dGrid,dim1dBlock>>>(Q_Lz, Q_Rz, F_z, nx_s, ny_s, nz_s, n_ghost, gama, 2, n_fields);
    #endif //EXACT
    #ifdef ROE
    Calculate_Roe_Fluxes_CUDA<<<dim1dGrid,dim1dBlock>>>(Q_Lx, Q_Rx, F_x, nx_s, ny_s, nz_s, n_ghost, gama, etah_x, 0, n_fields);
    Calculate_Roe_Fluxes_CUDA<<<dim1dGrid,dim1dBlock>>>(Q_Ly, Q_Ry, F_y, nx_s, ny_s, nz_s, n_ghost, gama, etah_y, 1, n_fields);
    Calculate_Roe_Fluxes_CUDA<<<dim1dGrid,dim1dBlock>>>(Q_Lz, Q_Rz, F_z, nx_s, ny_s, nz_s, n_ghost, gama, etah_z, 2, n_fields);
    #endif //ROE
    #ifdef HLLC
    Calculate_HLLC_Fluxes_CUDA<<<dim1dGrid,dim1dBlock>>>(Q_Lx, Q_Rx, F_x, nx_s, ny_s, nz_s, n_ghost, gama, etah_x, 0, n_fields);
    Calculate_HLLC_Fluxes_CUDA<<<dim1dGrid,dim1dBlock>>>(Q_Ly, Q_Ry, F_y, nx_s, ny_s, nz_s, n_ghost, gama, etah_y, 1, n_fields);
    Calculate_HLLC_Fluxes_CUDA<<<dim1dGrid,dim1dBlock>>>(Q_Lz, Q_Rz, F_z, nx_s, ny_s, nz_s, n_ghost, gama, etah_z, 2, n_fields);
    #endif //HLLC
    CudaCheckError();


    // Step 3: Update the conserved variables half a timestep
    Update_Conserved_Variables_3D_half<<<dim1dGrid,dim1dBlock>>>(dev_conserved, dev_conserved_half, F_x, F_y, F_z, nx_s, ny_s, nz_s, n_ghost, dx, dy, dz, 0.5*dt, gama, n_fields, dens_floor);
    CudaCheckError();


    // Step 4: Construct left and right interface values using updated conserved variables
    #ifdef PCM
    PCM_Reconstruction_3D<<<dim1dGrid,dim1dBlock>>>(dev_conserved_half, Q_Lx, Q_Rx, Q_Ly, Q_Ry, Q_Lz, Q_Rz, nx_s, ny_s, nz_s, n_ghost, gama);
    #endif
    #ifdef PLMP
    PLMP_cuda<<<dim1dGrid,dim1dBlock>>>(dev_conserved_half, Q_Lx, Q_Rx, nx_s, ny_s, nz_s, n_ghost, dx, dt, gama, 0, n_fields);
    PLMP_cuda<<<dim1dGrid,dim1dBlock>>>(dev_conserved_half, Q_Ly, Q_Ry, nx_s, ny_s, nz_s, n_ghost, dy, dt, gama, 1, n_fields);
    PLMP_cuda<<<dim1dGrid,dim1dBlock>>>(dev_conserved_half, Q_Lz, Q_Rz, nx_s, ny_s, nz_s, n_ghost, dz, dt, gama, 2, n_fields);
    #endif //PLMP
    #ifdef PLMC
    PLMC_cuda<<<dim1dGrid,dim1dBlock>>>(dev_conserved_half, Q_Lx, Q_Rx, nx_s, ny_s, nz_s, n_ghost, dx, dt, gama, 0, n_fields);
    PLMC_cuda<<<dim1dGrid,dim1dBlock>>>(dev_conserved_half, Q_Ly, Q_Ry, nx_s, ny_s, nz_s, n_ghost, dy, dt, gama, 1, n_fields);
    PLMC_cuda<<<dim1dGrid,dim1dBlock>>>(dev_conserved_half, Q_Lz, Q_Rz, nx_s, ny_s, nz_s, n_ghost, dz, dt, gama, 2, n_fields);
    #endif
    #ifdef PPMP
    PPMP_cuda<<<dim1dGrid,dim1dBlock>>>(dev_conserved_half, Q_Lx, Q_Rx, nx_s, ny_s, nz_s, n_ghost, dx, dt, gama, 0, n_fields);
    PPMP_cuda<<<dim1dGrid,dim1dBlock>>>(dev_conserved_half, Q_Ly, Q_Ry, nx_s, ny_s, nz_s, n_ghost, dy, dt, gama, 1, n_fields);
    PPMP_cuda<<<dim1dGrid,dim1dBlock>>>(dev_conserved_half, Q_Lz, Q_Rz, nx_s, ny_s, nz_s, n_ghost, dz, dt, gama, 2, n_fields);
    #endif //PPMP
    #ifdef PPMC
    PPMC_cuda<<<dim1dGrid,dim1dBlock>>>(dev_conserved_half, Q_Lx, Q_Rx, nx_s, ny_s, nz_s, n_ghost, dx, dt, gama, 0, n_fields);
    PPMC_cuda<<<dim1dGrid,dim1dBlock>>>(dev_conserved_half, Q_Ly, Q_Ry, nx_s, ny_s, nz_s, n_ghost, dy, dt, gama, 1, n_fields);
    PPMC_cuda<<<dim1dGrid,dim1dBlock>>>(dev_conserved_half, Q_Lz, Q_Rz, nx_s, ny_s, nz_s, n_ghost, dz, dt, gama, 2, n_fields);
    #endif //PPMC
    CudaCheckError();


    #ifdef H_CORRECTION
    // Step 4.5: Calculate eta values for H correction
    calc_eta_x_3D<<<dim1dGrid,dim1dBlock>>>(Q_Lx, Q_Rx, eta_x, nx_s, ny_s, nz_s, n_ghost, gama);
    calc_eta_y_3D<<<dim1dGrid,dim1dBlock>>>(Q_Ly, Q_Ry, eta_y, nx_s, ny_s, nz_s, n_ghost, gama);
    calc_eta_z_3D<<<dim1dGrid,dim1dBlock>>>(Q_Lz, Q_Rz, eta_z, nx_s, ny_s, nz_s, n_ghost, gama);
    CudaCheckError();
    // and etah values for each interface
    calc_etah_x_3D<<<dim1dGrid,dim1dBlock>>>(eta_x, eta_y, eta_z, etah_x, nx_s, ny_s, nz_s, n_ghost);
    calc_etah_y_3D<<<dim1dGrid,dim1dBlock>>>(eta_x, eta_y, eta_z, etah_y, nx_s, ny_s, nz_s, n_ghost);
    calc_etah_z_3D<<<dim1dGrid,dim1dBlock>>>(eta_x, eta_y, eta_z, etah_z, nx_s, ny_s, nz_s, n_ghost);
    CudaCheckError();
    #endif //H_CORRECTION


    // Step 5: Calculate the fluxes again
    #ifdef EXACT
    Calculate_Exact_Fluxes_CUDA<<<dim1dGrid,dim1dBlock>>>(Q_Lx, Q_Rx, F_x, nx_s, ny_s, nz_s, n_ghost, gama, 0, n_fields);
    Calculate_Exact_Fluxes_CUDA<<<dim1dGrid,dim1dBlock>>>(Q_Ly, Q_Ry, F_y, nx_s, ny_s, nz_s, n_ghost, gama, 1, n_fields);
    Calculate_Exact_Fluxes_CUDA<<<dim1dGrid,dim1dBlock>>>(Q_Lz, Q_Rz, F_z, nx_s, ny_s, nz_s, n_ghost, gama, 2, n_fields);
    #endif //EXACT
    #ifdef ROE
    Calculate_Roe_Fluxes_CUDA<<<dim1dGrid,dim1dBlock>>>(Q_Lx, Q_Rx, F_x, nx_s, ny_s, nz_s, n_ghost, gama, etah_x, 0, n_fields);
    Calculate_Roe_Fluxes_CUDA<<<dim1dGrid,dim1dBlock>>>(Q_Ly, Q_Ry, F_y, nx_s, ny_s, nz_s, n_ghost, gama, etah_y, 1, n_fields);
    Calculate_Roe_Fluxes_CUDA<<<dim1dGrid,dim1dBlock>>>(Q_Lz, Q_Rz, F_z, nx_s, ny_s, nz_s, n_ghost, gama, etah_z, 2, n_fields);
    #endif //ROE
    #ifdef HLLC
    Calculate_HLLC_Fluxes_CUDA<<<dim1dGrid,dim1dBlock>>>(Q_Lx, Q_Rx, F_x, nx_s, ny_s, nz_s, n_ghost, gama, etah_x, 0, n_fields);
    Calculate_HLLC_Fluxes_CUDA<<<dim1dGrid,dim1dBlock>>>(Q_Ly, Q_Ry, F_y, nx_s, ny_s, nz_s, n_ghost, gama, etah_y, 1, n_fields);
    Calculate_HLLC_Fluxes_CUDA<<<dim1dGrid,dim1dBlock>>>(Q_Lz, Q_Rz, F_z, nx_s, ny_s, nz_s, n_ghost, gama, etah_z, 2, n_fields);
    #endif //HLLC
    CudaCheckError();


    // Step 6: Update the conserved variable array
    Update_Conserved_Variables_3D<<<dim1dGrid,dim1dBlock>>>(dev_conserved, F_x, F_y, F_z, nx_s, ny_s, nz_s, x_off_s, y_off_s, z_off_s, n_ghost, dx, dy, dz, xbound, ybound, zbound, dt, gama, n_fields, dens_floor);
    CudaCheckError();

    #ifndef GRAVITY_CPU
    #ifdef DE
    Sync_Energies_3D<<<dim1dGrid,dim1dBlock>>>(dev_conserved, nx_s, ny_s, nz_s, n_ghost, gama, n_fields);
    CudaCheckError();

    #ifdef TEMPERATURE_FLOOR
    Apply_Temperature_Floor<<<dim1dGrid,dim1dBlock>>>(dev_conserved, nx_s, ny_s, nz_s, n_ghost, n_fields, temp_floor );
    CudaCheckError();
    #endif //TEMPERATURE_FLOOR
    #endif //DE
    #endif //GRAVITY_CPU

    // Apply cooling
    #ifdef COOLING_GPU
    cooling_kernel<<<dim1dGrid,dim1dBlock>>>(dev_conserved, nx_s, ny_s, nz_s, n_ghost, n_fields, dt, gama, dev_dt_array);
    CudaCheckError();
    #endif

    // Step 7: Calculate the next time step
    Calc_dt_3D<<<dim1dGrid,dim1dBlock>>>(dev_conserved, nx_s, ny_s, nz_s, n_ghost, dx, dy, dz, dev_dti_array, gama);
    CudaCheckError();

    // copy the updated conserved variable array back to the CPU
    CudaSafeCall( cudaMemcpy(tmp2, dev_conserved, n_fields*BLOCK_VOL*sizeof(Real), cudaMemcpyDeviceToHost) );

    // copy the updated conserved variable array from the buffer into the host_conserved array on the CPU
    host_return_block_3D(nx, ny, nz, nx_s, ny_s, nz_s, n_ghost, block, block1_tot, block2_tot, block3_tot, remainder1, remainder2, remainder3, BLOCK_VOL, host_conserved1, buffer, n_fields);

    // copy the dti array onto the CPU
    CudaSafeCall( cudaMemcpy(host_dti_array, dev_dti_array, ngrid*sizeof(Real), cudaMemcpyDeviceToHost) );
    // find maximum inverse timestep from CFL condition
    for (int i=0; i<ngrid; i++) {
      max_dti = fmax(max_dti, host_dti_array[i]);
    }
    #ifdef COOLING_GPU
    // copy the dt array from cooling onto the CPU
    CudaSafeCall( cudaMemcpy(host_dt_array, dev_dt_array, ngrid*sizeof(Real), cudaMemcpyDeviceToHost) );
    // find maximum inverse timestep from cooling time
    for (int i=0; i<ngrid; i++) {
      min_dt = fmin(min_dt, host_dt_array[i]);
    }
    if (min_dt < C_cfl/max_dti) {
      max_dti = C_cfl/min_dt;
    }
    #endif

    // add one to the counter
    block++;

  }

  // free CPU memory
  free(host_dti_array);
  if (block_tot > 1) free(buffer);
  #ifdef COOLING_GPU
  free(host_dt_array);
  #endif

  // // free the GPU memory
  // cudaFree(dev_conserved);
  // cudaFree(dev_conserved_half);
  // cudaFree(Q_Lx);
  // cudaFree(Q_Rx);
  // cudaFree(Q_Ly);
  // cudaFree(Q_Ry);
  // cudaFree(Q_Lz);
  // cudaFree(Q_Rz);
  // cudaFree(F_x);
  // cudaFree(F_y);
  // cudaFree(F_z);
  // cudaFree(eta_x);
  // cudaFree(eta_y);
  // cudaFree(eta_z);
  // cudaFree(etah_x);
  // cudaFree(etah_y);
  // cudaFree(etah_z);
  // cudaFree(dev_dti_array);
  // #ifdef COOLING_GPU
  // cudaFree(dev_dt_array);
  // #endif

  // return the maximum inverse timestep
  return max_dti;

}


__global__ void Update_Conserved_Variables_3D_half(Real *dev_conserved, Real *dev_conserved_half, Real *dev_F_x, Real *dev_F_y,  Real *dev_F_z, int nx, int ny, int nz, int n_ghost, Real dx, Real dy, Real dz, Real dt, Real gamma, int n_fields, Real dens_floor )
{
  Real dtodx = dt/dx;
  Real dtody = dt/dy;
  Real dtodz = dt/dz;
  int n_cells = nx*ny*nz;

  // get a global thread ID
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  int zid = tid / (nx*ny);
  int yid = (tid - zid*nx*ny) / nx;
  int xid = tid - zid*nx*ny - yid*nx;
  int id = xid + yid*nx + zid*nx*ny;

  int imo = xid-1 + yid*nx + zid*nx*ny;
  int jmo = xid + (yid-1)*nx + zid*nx*ny;
  int kmo = xid + yid*nx + (zid-1)*nx*ny;

  #ifdef DE
  Real d, d_inv, vx, vy, vz;
  Real vx_imo, vx_ipo, vy_jmo, vy_jpo, vz_kmo, vz_kpo, P;
  // Real GE;
  int ipo, jpo, kpo;
  #endif

  #ifdef ENERGY_FLOOR
  Real E, Ek;
  #endif

  // threads corresponding to all cells except outer ring of ghost cells do the calculation
  if (xid > 0 && xid < nx-1 && yid > 0 && yid < ny-1 && zid > 0 && zid < nz-1)
  {
    #ifdef DE
    d  =  dev_conserved[            id];
    d_inv = 1.0 / d;
    vx =  dev_conserved[1*n_cells + id] * d_inv;
    vy =  dev_conserved[2*n_cells + id] * d_inv;
    vz =  dev_conserved[3*n_cells + id] * d_inv;
    P  = (dev_conserved[4*n_cells + id] - 0.5*d*(vx*vx + vy*vy + vz*vz)) * (gamma - 1.0);
    // GE = fmin(dev_conserved[(n_fields-1)*n_cells + id], 1e-6);
    // if (d < 0.0 || d != d) printf("Negative density before half step update.\n");
    if (P < 0.0) P  = dev_conserved[(n_fields-1)*n_cells + id] * (gamma - 1.0);
    if (P < 0.0) printf("%d Negative pressure before final update.\n", id);
    ipo = xid+1 + yid*nx + zid*nx*ny;
    jpo = xid + (yid+1)*nx + zid*nx*ny;
    kpo = xid + yid*nx + (zid+1)*nx*ny;
    vx_imo = dev_conserved[1*n_cells + imo] / dev_conserved[imo];
    vx_ipo = dev_conserved[1*n_cells + ipo] / dev_conserved[ipo];
    vy_jmo = dev_conserved[2*n_cells + jmo] / dev_conserved[jmo];
    vy_jpo = dev_conserved[2*n_cells + jpo] / dev_conserved[jpo];
    vz_kmo = dev_conserved[3*n_cells + kmo] / dev_conserved[kmo];
    vz_kpo = dev_conserved[3*n_cells + kpo] / dev_conserved[kpo];
    #endif

    // update the conserved variable array
    dev_conserved_half[            id] = dev_conserved[            id]
                                       + dtodx * (dev_F_x[            imo] - dev_F_x[            id])
                                       + dtody * (dev_F_y[            jmo] - dev_F_y[            id])
                                       + dtodz * (dev_F_z[            kmo] - dev_F_z[            id]);
    dev_conserved_half[  n_cells + id] = dev_conserved[  n_cells + id]
                                       + dtodx * (dev_F_x[  n_cells + imo] - dev_F_x[  n_cells + id])
                                       + dtody * (dev_F_y[  n_cells + jmo] - dev_F_y[  n_cells + id])
                                       + dtodz * (dev_F_z[  n_cells + kmo] - dev_F_z[  n_cells + id]);
    dev_conserved_half[2*n_cells + id] = dev_conserved[2*n_cells + id]
                                       + dtodx * (dev_F_x[2*n_cells + imo] - dev_F_x[2*n_cells + id])
                                       + dtody * (dev_F_y[2*n_cells + jmo] - dev_F_y[2*n_cells + id])
                                       + dtodz * (dev_F_z[2*n_cells + kmo] - dev_F_z[2*n_cells + id]);
    dev_conserved_half[3*n_cells + id] = dev_conserved[3*n_cells + id]
                                       + dtodx * (dev_F_x[3*n_cells + imo] - dev_F_x[3*n_cells + id])
                                       + dtody * (dev_F_y[3*n_cells + jmo] - dev_F_y[3*n_cells + id])
                                       + dtodz * (dev_F_z[3*n_cells + kmo] - dev_F_z[3*n_cells + id]);
    dev_conserved_half[4*n_cells + id] = dev_conserved[4*n_cells + id]
                                       + dtodx * (dev_F_x[4*n_cells + imo] - dev_F_x[4*n_cells + id])
                                       + dtody * (dev_F_y[4*n_cells + jmo] - dev_F_y[4*n_cells + id])
                                       + dtodz * (dev_F_z[4*n_cells + kmo] - dev_F_z[4*n_cells + id]);
    #ifdef SCALAR
    for (int i=0; i<NSCALARS; i++) {
      dev_conserved_half[(5+i)*n_cells + id] = dev_conserved[(5+i)*n_cells + id]
                                         + dtodx * (dev_F_x[(5+i)*n_cells + imo] - dev_F_x[(5+i)*n_cells + id])
                                         + dtody * (dev_F_y[(5+i)*n_cells + jmo] - dev_F_y[(5+i)*n_cells + id])
                                         + dtodz * (dev_F_z[(5+i)*n_cells + kmo] - dev_F_z[(5+i)*n_cells + id]);
    }
    #endif
    #ifdef DE
    dev_conserved_half[(n_fields-1)*n_cells + id] = dev_conserved[(n_fields-1)*n_cells + id]
                                       + dtodx * (dev_F_x[(n_fields-1)*n_cells + imo] - dev_F_x[(n_fields-1)*n_cells + id])
                                       + dtody * (dev_F_y[(n_fields-1)*n_cells + jmo] - dev_F_y[(n_fields-1)*n_cells + id])
                                       + dtodz * (dev_F_z[(n_fields-1)*n_cells + kmo] - dev_F_z[(n_fields-1)*n_cells + id])
                                       + 0.5*P*(dtodx*(vx_imo-vx_ipo) + dtody*(vy_jmo-vy_jpo) + dtodz*(vz_kmo-vz_kpo));
    #endif

    #ifdef DENSITY_FLOOR
    if ( dev_conserved[            id] < dens_floor ){
      printf("###Thread density change  %f -> %f \n", dev_conserved[            id], dens_floor );
      dev_conserved[            id] = dens_floor;
    }
    #endif

    #ifdef DE
    #ifdef ENERGY_FLOOR
    d  =  dev_conserved[            id];
    d_inv = 1.0 / d;
    vx =  dev_conserved[1*n_cells + id] * d_inv;
    vy =  dev_conserved[2*n_cells + id] * d_inv;
    vz =  dev_conserved[3*n_cells + id] * d_inv;
    E = dev_conserved[4*n_cells + id];
    Ek = 0.5 * d * ( vx*vx + vy*vy + vz*vz );
    if (dev_conserved_half[(n_fields-1)*n_cells + id] < 0 ) dev_conserved_half[(n_fields-1)*n_cells + id] = 1e-5;
    if ( E - Ek < 0 ){
      dev_conserved[4*n_cells + id] = Ek + dev_conserved_half[(n_fields-1)*n_cells + id];
      printf("###Thread Energy change  %f -> %f \n", E, dev_conserved[4*n_cells + id] );
    }

    #endif
    #endif
    //if (dev_conserved_half[id] < 0.0 || dev_conserved_half[id] != dev_conserved_half[id] || dev_conserved_half[4*n_cells+id] < 0.0 || dev_conserved_half[4*n_cells+id] != dev_conserved_half[4*n_cells+id]) {
      //printf("%3d %3d %3d Thread crashed in half step update. d: %e E: %e\n", xid, yid, zid, dev_conserved_half[id], dev_conserved_half[4*n_cells+id]);
    //}

  }

}




#endif //VL
#endif //CUDA
