
#include <hip/hip_runtime.h>
#ifdef GRAVITY
#ifdef POTENTIAL_CUFFT

#include "potential_CUFFT_3D.h"


Potential_CUFFT_3D::Potential_CUFFT_3D( void ){}

void Potential_CUFFT_3D::Initialize( Grav3D Grav){

  Lbox_x = Grav.Lbox_x;
  Lbox_y = Grav.Lbox_y;
  Lbox_z = Grav.Lbox_z;

  nx_total = Grav.nx_total;
  ny_total = Grav.ny_total;
  nz_total = Grav.nz_total;

  nx_local = Grav.nx_local;
  ny_local = Grav.ny_local;
  nz_local = Grav.nz_local;

  dx = Grav.dx;
  dy = Grav.dy;
  dz = Grav.dz;

  chprintf( " Using Poisson Solver: CUFFT\n");
chprintf( "  CUFFT: L[ %f %f %f ] N[ %d %d %d ] dx[ %f %f %f ]\n", Lbox_x, Lbox_y, Lbox_z, nx_local, ny_local, nz_local, dx, dy, dz );



}




#endif //POTENTIAL_CUFFT
#endif //GRAVITY
