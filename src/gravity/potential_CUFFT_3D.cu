
#include <hip/hip_runtime.h>
#ifdef GRAVITY
#ifdef POTENTIAL_CUFFT

#include "potential_CUFFT_3D.h"


Potential_CUFFT_3D::Potential_CUFFT_3D( void ){}

void Potential_CUFFT_3D::Initialize( Grav3D Grav){

  Lbox_x = Grav.Lbox_x;
  Lbox_y = Grav.Lbox_y;
  Lbox_z = Grav.Lbox_z;

  nx_total = Grav.nx_total;
  ny_total = Grav.ny_total;
  nz_total = Grav.nz_total;

  nx_local = Grav.nx_local;
  ny_local = Grav.ny_local;
  nz_local = Grav.nz_local;

  dx = Grav.dx;
  dy = Grav.dy;
  dz = Grav.dz;

  n_cells_local = nx_local*ny_local*nz_local;
  n_cells_total = nx_total*ny_total*nz_total;
  chprintf( " Using Poisson Solver: CUFFT\n");
  chprintf( "  CUFFT: L[ %f %f %f ] N[ %d %d %d ] dx[ %f %f %f ]\n", Lbox_x, Lbox_y, Lbox_z, nx_local, ny_local, nz_local, dx, dy, dz );

  AllocateMemory_CPU();

  chprintf( "  CUFFT: Creating FFT plan...\n");
  cufftPlan3d( &plan_cufft_fwd,  nz_local, ny_local,  nx_local, CUFFT_D2Z);
  cufftPlan3d( &plan_cufft_bwd,  nz_local, ny_local,  nx_local, CUFFT_Z2D);


}

void Potential_CUFFT_3D::AllocateMemory_CPU( void ){
  F.output_h = (Real *) malloc(n_cells_local*sizeof(Real));

}

void Potential_CUFFT_3D::AllocateMemory_GPU( void ){

  cudaMalloc( (void**)&F.transform_d, n_cells_local*sizeof(Complex_cufft));
  cudaMalloc( (void**)&F.input_d, n_cells_local*sizeof(Real_cufft));
  cudaMalloc( (void**)&F.output_d, n_cells_local*sizeof(Real_cufft));

}

void Potential_CUFFT_3D::FreeMemory_GPU( void ){
  cudaFree( F.input_d );
  cudaFree( F.output_d );
  cudaFree( F.transform_d );
}

void Potential_CUFFT_3D::Copy_Input( Grav3D &Grav ){
  cudaMemcpy( F.input_d, Grav.F.density_h, n_cells_local*sizeof(Real_cufft), cudaMemcpyHostToDevice );
}

void Potential_CUFFT_3D::Copy_Output( Grav3D &Grav ){

  cudaMemcpy( F.output_h, F.output_d, n_cells_local*sizeof(Real_cufft), cudaMemcpyDeviceToHost );
  // cudaMemcpy( F.output_h, F.input_d, n_cells_local*sizeof(Real_cufft), cudaMemcpyDeviceToHost );

  int id, id_pot;
  int i, k, j;
  for (k=0; k<nz_local; k++) {
    for (j=0; j<ny_local; j++) {
      for (i=0; i<nx_local; i++) {
        id = i + j*nx_local + k*nx_local*ny_local;
        id_pot = (i+N_GHOST_POTENTIAL) + (j+N_GHOST_POTENTIAL)*(nx_local+2*N_GHOST_POTENTIAL) + (k+N_GHOST_POTENTIAL)*(nx_local+2*N_GHOST_POTENTIAL)*(ny_local+2*N_GHOST_POTENTIAL);
        Grav.F.potential_h[id_pot] = F.output_h[id] / n_cells_local;
        // chprintf( "%f\n", Grav.F.potential_h[id]);
      }
    }
  }
}

void Potential_CUFFT_3D::Get_Potential( Grav3D &Grav ){

  cudaEvent_t start, stop;
  cudaEventCreate(&start);
  cudaEventCreate(&stop);
  cudaEventRecord(start);

  AllocateMemory_GPU();
  Copy_Input( Grav );

  cufftExecD2Z( plan_cufft_fwd, F.input_d, F.transform_d );
  cufftExecZ2D( plan_cufft_bwd, F.transform_d, F.output_d );
  Copy_Output( Grav );

  FreeMemory_GPU();

  cudaEventRecord(stop);
  cudaEventSynchronize(stop);
  float milliseconds = 0;
  cudaEventElapsedTime(&milliseconds, start, stop);
  chprintf( " CUFFT: Potential Time = %f   msecs\n", milliseconds);
}




#endif //POTENTIAL_CUFFT
#endif //GRAVITY
