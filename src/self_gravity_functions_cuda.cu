
#include <hip/hip_runtime.h>
#ifdef CUDA
#ifdef SELF_GRAVITY

#include<math.h>
#include<cuda.h>
#include"global_cuda.h"

//void CopyField_Host_To_Device( Real *field_h, Real *field_d, int n_cells ){
void CopyField_Host_To_Device( Real *field_h, Real *field_d, long n_cells ){
  CudaSafeCall( cudaMemcpy( field_d, field_h , n_cells*sizeof(Real), cudaMemcpyHostToDevice) );
}

#endif //SELF_GRAVITY
#endif
